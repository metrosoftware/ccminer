#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

 extern "C"
 {
 #include "sph/sph_shavite.h"
 #include "sph/sph_simd.h"
 #include "sph/sph_keccak.h"
 }
 #include "miner.h"
 
 
 #include "cuda_helper.h"
 
 extern void keccak256_cpu_init(int thr_id, uint32_t threads);
 extern void keccak256_setBlock_M(int thr_id, void *pdata,const void *ptarget);
 extern void keccak256_cpu_hash_M(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce);
 
 // CPU Hash
 void keccak256_general_hash(void *state, const void *input)
 {
	 sph_keccak_context ctx_keccak;
 
	 uint32_t hash[16];
 
	 sph_keccak256_init(&ctx_keccak);
	 sph_keccak256 (&ctx_keccak, input, len);
	 sph_keccak256_close(&ctx_keccak, (void*) hash);
 
	 memcpy(state, hash, 32);
 }

 void keccak256_metro_hash(void *state, const void *input)
 {
	 keccak256_general_hash(state, input, 98);
 }

 extern void setnounce(uint32_t *pdata, uint32_t nounce) {
    pdata[23] = (pdata[23] & 0xFFFFU) | ((nounce & 0xFFFFU) << 16);
    pdata[24] = (nounce & 0xFFFF0000U) >> 16;
}

extern uint32_t getnounce(uint32_t *pdata) {
    return ((pdata[23] >> 16) & 0xFFFFU) + ((pdata[24] & 0xFFFFU) << 16);
}

 extern int scanhash_keccak256_metro(int thr_id, uint32_t *pdata,
	 uint32_t *ptarget, uint32_t max_nonce,
	 uint32_t *hashes_done)
 {
	 static THREAD uint32_t *h_nounce = nullptr;
 
	 const uint32_t first_nonce = getnounce(pdata);
	 uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	 uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	 uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;
 
 
	 if (opt_benchmark)
		 ptarget[7] = 0x0002;
 
	 static THREAD volatile bool init = false;
	 if(!init)
	 {
		 if(throughputmax == intensity)
			 applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		 CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		 CUDA_SAFE_CALL(hipDeviceReset());
		 CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		 CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		 CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		 CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, 2 * sizeof(uint32_t)));
		 keccak256_cpu_init(thr_id, (int)throughputmax);
 //		CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, 2 * sizeof(uint32_t)));
		 mining_has_stopped[thr_id] = false;
		 init = true;
	 }
 
	 uint32_t endiandata[25];
     memcpy(&endiandata, pdata, 98);
 	 memset(((unsigned char*)&endiandata) + 98, 0, 2);

	 keccak256_setBlock_M(thr_id, (void*)endiandata, ptarget);
 
	 do {
 
		 keccak256_cpu_hash_M(thr_id, (int) throughput, getnounce(pdata), h_nounce);
		 if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		 if(h_nounce[0] != UINT32_MAX)
		 {
			 uint32_t Htarg = ptarget[7];
			 uint32_t vhash64[8]={0};
			 if(opt_verify){
	             *((uint32_t*)((unsigned char*)&endiandata[23] + 2)) = h_nounce[0];
                 keccak256_metro_hash(vhash64, endiandata);
			 }
			 if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			 {
				 int res = 1;
				 // check if there was some other ones...
				 *hashes_done = getnounce(pdata) - first_nonce + throughput;
				 if (h_nounce[1] != 0xffffffff)
				 {
					 if(opt_verify){
	                 *((uint32_t*)((unsigned char*)&endiandata[23] + 2)) = h_nounce[1];
					 keccak256_metro_hash(vhash64, endiandata);
 
					 }
					 if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					 {
						 pdata[26] = h_nounce[1];
						 res++;
						 if (opt_benchmark)
							 applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_nounce[1]);
					 }
					 else
					 {
						 if (vhash64[7] != Htarg)
						 {
							 applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_nounce[1]);
						 }
					 }
				 }
				 setnounce(pdata, h_nounce[0]);
				 if (opt_benchmark)
					 applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_nounce[0]);
				 return res;
			 }
			 else
			 {
				 if (vhash64[7] != Htarg)
				 {
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_nounce[0]);
				}
			 }
		 }
 
		 setnounce(pdata, getnounce(pdata) + throughput); CUDA_SAFE_CALL(hipGetLastError());
	 } while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(getnounce(pdata)) + (uint64_t)throughput)));
	 *hashes_done = getnounce(pdata) - first_nonce ;
	 return 0;
 }
 